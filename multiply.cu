#include <hip/hip_runtime.h>


__global__ void __multiply__ (const float *a, float *b, int n) {
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        b[i] *= a[i];
    }
}

extern "C" void launch_multiply(const float *a, float *b, int n) {
    float *a_gpu = nullptr;
    float *b_gpu = nullptr;

    // Allocate memory on GPU
    hipMalloc((void**)&a_gpu, n * sizeof(float));
    hipMalloc((void**)&b_gpu, n * sizeof(float));

    // Copy data from host to device
    hipMemcpy(a_gpu, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    __multiply__<<<blocks, threadsPerBlock>>>(a_gpu, b_gpu, n);

    // Error check and sync
    hipGetLastError();
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(b, b_gpu, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(a_gpu);
    hipFree(b_gpu);
}
